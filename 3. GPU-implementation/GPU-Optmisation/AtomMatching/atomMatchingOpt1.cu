#include "hip/hip_runtime.h"
#include "atomMatchingOpt1.h"
#define NC 3




__global__ void atomMatchingOpt1(float* c, const float *a, const float *b,
	const int NA, const int NB, const int NMax, float threshold);
hipError_t atomMatchingWithCuda1(float* c, const float *a, const float *b,
			const int NA, const int NB, const int NMax, float threshold,
			string outputFileName);




/**
Compare every element in each row of a to every element in each row of b.
The match distances is stored in c.
refer to the documentation for explanation

input:
  array of float c: containing result
  array of float a: containing coordinates of molecule A
  array of float b: containing coordinates of other molecules to compare with A
  const int NA: number of atoms in molecule A
  const int NB: number of atoms in each molecule in B
  const int NMax: number of molecules in B
  float threshold: acceptable difference for coordinates to be the same

output:
  void
**/
__global__ void atomMatchingOpt1(float* c, const float *a,
		const float *b,const int NA, const int NB, const int NMax, float threshold){

	int  tempA, tempB, tid;
	float result;

	tid= blockIdx.x*blockDim.x+threadIdx.x;

  //Total number of threads to compare 1 molecule of NA atoms and NMax molecule
  // of NB atoms is NA*NB*NMax
	if (tid <NB*NA*NMax) {

    //First element in float *a for each thread to look at.
    tempA = (( tid / NB ) % NA ) * NA;
    //First element in float *b for each thread to look at.
    tempB = ( ( tid / (NA*NB) ) *NB + (tid% NB) )*NB;

		result = 0;
    //Local memory for each thread containg NB boolean value. (NC = NB)
		bool array[NC] = {0};

    //looping through all elements in float *a and float *b
		for (int k =0; k<NA; k++) {
			for (int t =0; t<NB; t++) {
        //first, check a b's element has not be matched with an a's element previously
        // then, check whether the a's and b's elements are similiry within a threshold
				if (!array[(tempB+t)%NB] && (abs(a[tempA+k]-b[tempB+t])<= threshold)) {
          // increase the count
					result = result  +1;
          //signal that the b's element is now matched with one a's element
					array[(tempB+t)%NB] = true;
          //stop the current loop
					break;
				}
			}
		}
		c[tid] = result;
	}
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t atomMatchingWithCuda1(float* c, const float *a, const float *b,
		const int NA, const int NB, const int NMax, float threshold,
		string outputFileName)      {

	float *dev_a = 0;
	float *dev_b = 0;
	float *dev_c = 0;
	hipError_t cudaStatus;
	hipEvent_t start, stop;
	float milliseconds;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

  //Start timer
  cudaStatus = hipEventCreate(&start);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipEventCreate(& start) failed!\n");
		goto Error;
	}

	cudaStatus =   hipEventCreate(&stop);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipEventCreate(& stop) failed!\n");
		goto Error;
	}

	cudaStatus = hipEventRecord(start);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipEventRecord(start) failed!\n");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, NA*NB*NMax * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, NA*NA * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, NB*NB * NMax *sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, NA * NA * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed for dev_a!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, NB  * NB *NMax *sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed! for dev_b");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	//int block = (NA*NB*NMax)/128+1;
	atomMatchingOpt1<<< 128, (NA*NB*NMax)/128+1>>>(dev_c, dev_a, dev_b, NA, NB, NMax, threshold);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}


	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, NA*NB*NMax * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed for dev_c!\n");
		goto Error;
	}


  //stop timer
  cudaStatus =   hipEventRecord(stop);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipEventRecord(start) failed!\n");
		goto Error;
	}

  // hipDeviceSynchronize waits for the kernel to finish, and returns
  // any errors encountered during the launch.
  cudaStatus = hipDeviceSynchronize();
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
    goto Error;
  }

  //Get elapsed times
  cudaStatus = hipEventElapsedTime(&milliseconds, start, stop);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipEventElapsedTime failed!\n");
		goto Error;
	}

  //Write result into the specified file
  	writeResult2File (NA,NB,NMax,milliseconds,"miilliseconds",outputFileName);


	Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);


	return cudaStatus;
}
