#include "hip/hip_runtime.h"
#include "atomMatchingOpt5.h"
using namespace std;

__global__ void atomMatchingOpt5(float* c, const float *a, const float *b,
	const int NA, const int NB, const int NMax, float threshold);
hipError_t atomMatchingWithCuda5(float* c, const float *a, const float *b,
			const int NA, const int NB, const int NMax, float threshold,
			string outputFileName);

/**

int main() {

    const int NA = 2;
    const int NB = 3;
    const int NMax = 1;
    hipError_t cudaStatus;

    float A[NA*NA] = {2.0,0.0,4.0,1.0};
    float B[NB*NB*NMax] = {1.5,0.5,0.0,  4.2,0.7,1.0,  10.0,12.0,19.0};//{2.0,1.0,0.0,  4.0,0.5,2.0,  3.0,0.0,1.0,  1.5,0.5,0.0,  4.2,0.7,1.0,  10.0,12.0,19.0};
    float C[NB*NA*NMax];


		cudaStatus = atomMatchingWithCuda5(C, A,B,NA,NB, NMax,0.5f,"atomMatchingWithCuda5.txt");

    for (int i = 0; i< NMax;i++) {
    for (int j =0; j<NB*NA; j++)  printf("%.2f ", C[NB*NA*i+j]);
    cout << endl;
    }

		testFunction (A, B, NA, NB,  NMax, 500, 4000, 500,0.5f, "atomMatchingOpt5.txt",&atomMatchingWithCuda5);
    testFunction (A, B, NA, NB,  NMax, 10000, 100000, 10000,0.5f, "atomMatchingOpt5.txt",&atomMatchingWithCuda5);
    if (cudaStatus != hipSuccess) {
      fprintf(stderr, "addWithCuda failed!");
      return 1;
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}


**/

/**
Compare every element in each row of a to every element in each row of b.
The match distances is stored in c.
refer to the documentation for explanation

input:
  array of float c: containing result
  array of float a: containing coordinates of molecule A
  array of float b: containing coordinates of other molecules to compare with A
  const int NA: number of atoms in molecule A
  const int NB: number of atoms in each molecule in B
  const int NMax: number of molecules in B
  float threshold: acceptable difference for coordinates to be the same

output:
  void
**/


__global__ void atomMatchingOpt5(float* c, const float *a,
		const float *b,const int NA, const int NB, const int NMax, float threshold){

	int  tempA, tempB, tid;
	float result;
	tid= blockIdx.x*blockDim.x+threadIdx.x;

	//Allocation of bool* array into local memory
	//Note: Due to limited shared memory, kernel call will fall with too large NMax
	extern __shared__ bool array[];
	if(tid <NA*NB*NB*NMax)   {
		array[tid]= false;
	}
	__syncthreads();

	//Total number of threads to compare 1 molecule of NA atoms and NMax molecule
	// of NB atoms is NA*NB*NMax
	if (tid <NB*NA*NMax) {
		//First element in float *a for each thread to look at.
		tempA = (( tid / NB ) % NA ) * NA;
		//First element in float *b for each thread to look at.
		tempB = ( ( tid / (NA*NB) ) *NB + (tid% NB) )*NB;
		result = 0;

		//looping through all elements in float *a and float *b
		for (int k =0; k<NA; k++) {
			for (int t =0; t<NB; t++) {
				//first, check a b's element has not be matched with an a's element previously
				// then, check whether the a's and b's elements are similiry within a threshold
				if (!array[tid*NB+t] && (abs(a[tempA+k]-b[tempB+t])<= threshold)) {
					// increase the count
					result = result  +1;
					//signal that the b's element is now matched with one a's element
					array[(tid*NB+t)%NB] = true;
					//stop the current loop
					break;
				}
			}
		}

		c[tid] = result;
	}
}


// Helper function for using CUDA to add vectors in parallel.
hipError_t atomMatchingWithCuda5(float* c, const float *a, const float *b,
		const int NA, const int NB, const int NMax, float threshold,
		string outputFileName)      {

	float *dev_a = 0;
	float *dev_b = 0;
	float *dev_c = 0;

	hipError_t cudaStatus;
	hipEvent_t start, stop;
	float milliseconds;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

  //Start timer
  cudaStatus = hipEventCreate(&start);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipEventCreate(& start) failed!\n");
		goto Error;
	}

	cudaStatus =   hipEventCreate(&stop);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipEventCreate(& stop) failed!\n");
		goto Error;
	}

	cudaStatus = hipEventRecord(start);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipEventRecord(start) failed!\n");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, NA*NB*NMax * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, NA*NA * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, NB*NB * NMax *sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, NA * NA * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed for dev_a!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, NB  * NB *NMax *sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed! for dev_b");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
		atomMatchingOpt5<<<128, (NA*NB*NMax)/128+1,NA*NB*NB*NMax*sizeof(bool) >>>(dev_c, dev_a, dev_b, NA, NB, NMax, threshold);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}


	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, NA*NB*NMax * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed for dev_c!\n");
		goto Error;
	}


  //stop timer
  cudaStatus =   hipEventRecord(stop);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipEventRecord(start) failed!\n");
		goto Error;
	}

  // hipDeviceSynchronize waits for the kernel to finish, and returns
  // any errors encountered during the launch.
  cudaStatus = hipDeviceSynchronize();
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
    goto Error;
  }

  //Get elapsed times
  cudaStatus = hipEventElapsedTime(&milliseconds, start, stop);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipEventElapsedTime failed!\n");
		goto Error;
	}

  //Write result into the specified file
    	writeResult2File (NA,NB,NMax,milliseconds,"miilliseconds",outputFileName);


	Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}
